#include "hip/hip_runtime.h"
﻿/*
 * EXAMPLE OF MATRIX-MATRIX MULTIPLICATION: CHAPTER 4
 */

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <chrono>
#include <iostream>
#include <iomanip>
using namespace std;
#define CHECK_ERROR(call) { \
	hipError_t err = call; \
	if (err != hipSuccess) { \
		printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
		exit(err); \
	} \
}

__global__
void matrixMulKernel(float* P, float* M, float* N, int Width) {
    int Col = blockDim.x * blockIdx.x + threadIdx.x;
    int Row = blockDim.y * blockIdx.y + threadIdx.y;

    // check that only the threads with both Row and Col values are in within range
    if (Row < Width && Col < Width) {
        float pValue = 0;
        for (int k = 0; k < Width; k++) {
            pValue += M[Row * Width + k] * N[k * Width + Col];
        }
        P[Row * Width + Col] = pValue;
    }
}


void matrixMul(float* h_P, float* h_M, float* h_N, int dim, int numThreads) {

    int size = (dim * dim) * sizeof(float);
    float* d_M, * d_N, * d_P;

    //1. Allocate global memory on the device for d_Pin and d_Pout
    // With this type of allocation it isn't possible acces using higher-dimensional indexing syntax
    // it need to linearize first.
    CHECK_ERROR(hipMalloc((void**)&d_M, size));
    CHECK_ERROR(hipMalloc((void**)&d_N, size));
    CHECK_ERROR(hipMalloc((void**)&d_P, size));    // assume square matricies

    // copy h_Pin to device memory
    hipMemcpy(d_M, h_M, size, hipMemcpyHostToDevice);
    hipMemcpy(d_N, h_N, size, hipMemcpyHostToDevice);

    //2. Kernel launch code - with 256 threads per block
    
    dim3 dimGrid(ceil(dim / numThreads), ceil(dim / numThreads), 1);
    dim3 dimBlock(16, 16, 1);
    matrixMulKernel << <dimGrid, dimBlock >> > (d_P, d_M, d_N, dim);

    //3. copy d_Pout from the device memory
    hipMemcpy(h_P, d_P, size, hipMemcpyDeviceToHost);

    // Free device vectors
    hipFree(d_M);
    hipFree(d_N);
    hipFree(d_P);
}

int main(int argc, char* argv[]) {
    int numThreadsList[] = { 32, 64, 128, 256, 512 };
    int numThreadsSize = sizeof(numThreadsList) / sizeof(numThreadsList[0]);

    int dimList[] = { 256, 512, 1024, 2048, 4096, 8192, 16384, 32768 };
    int dimSize = sizeof(dimList) / sizeof(dimList[0]);

    float* h_M, * h_N, * h_P;
    h_M = (float*)malloc(sizeof(float) * dimList[dimSize - 1] * dimList[dimSize - 1]);
    h_N = (float*)malloc(sizeof(float) * dimList[dimSize - 1] * dimList[dimSize - 1]);
    h_P = (float*)malloc(sizeof(float) * dimList[dimSize - 1] * dimList[dimSize - 1]);

    // fill M and N with random float numbers
    srand(time(NULL));
    for (int i = 0; i < dimSize; i++) {
        int dim = dimList[i];

        for (int j = 0; j < dim; j++) {
            for (int k = 0; k < dim; k++) {
                h_M[j * dim + k] = ((((float)rand() / (float)(RAND_MAX)) * 10));
                h_N[j * dim + k] = ((((float)rand() / (float)(RAND_MAX)) * 10));
            }
        }

        for (int j = 0; j < numThreadsSize; j++) {
            int numThreads = numThreadsList[j];

            auto start = chrono::steady_clock::now();
            // perform matrix multiplication
            matrixMul(h_P, h_M, h_N, dim, numThreads);
            auto end = chrono::steady_clock::now(); // Measure end time
            auto duration = chrono::duration_cast<chrono::duration<double, milli>>(end - start); // Calculate duration in milliseconds

            cout << "Para dim " << dim << " con " << numThreads << " threads, se tiene un tiempo de " << duration.count() << " ms" << endl;
        }

        cout << "--------------------------" << endl;
    }

    // Free host memory
    free(h_M);
    free(h_N);
    free(h_P);

    cout << "¡Multiplicación completada con éxito!" << endl;

    return 0;
}




